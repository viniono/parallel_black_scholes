#include "hip/hip_runtime.h"
#include "black.h"
#include "file_handling.h"
#include "time_util.h"
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>
#define THREADS_PER_BLOCK 64
#define MAX_LINE_SIZE 1024

typedef struct {
  double call_price;
  double put_price;
} option_price_t;
// __global__ void pricer(float a, float* x, float* y) {
__global__ void pricer(bs_inputs_t *blackScholes_inputs,
                       option_price_t *prices) {
  // Which index of the array should this thread use?
  size_t index = blockIdx.x * blockDim.x + threadIdx.x;

  // Unmarshall struct
  double K = blackScholes_inputs[index].K;
  double S = blackScholes_inputs[index].S;
  double r = blackScholes_inputs[index].r;
  double T = blackScholes_inputs[index].T;
  double sigma = blackScholes_inputs[index].sigma;
  // Compute prices parallell
  double d1 = D1(S, K, T, r, sigma);
  double d2 = D2(d1, sigma, T);
  prices[index].call_price = S * cdf(d1) - K * __expf(-r * T) * cdf(d2);
  prices[index].put_price = K * __expf(-r * T) - S + prices[index].call_price;
  // prices[index].put_price =
  //     BS_PUT(blackScholes_inputs[index].S, blackScholes_inputs[index].K,
  //            blackScholes_inputs[index].T, blackScholes_inputs[index].r,
  //            blackScholes_inputs[index].sigma);
  // prices[index].put_price =
  //     BS_CALL(blackScholes_inputs[index].S, blackScholes_inputs[index].K,
  //             blackScholes_inputs[index].T, blackScholes_inputs[index].r,
  //             blackScholes_inputs[index].sigma);
}

int main() {

  int N;
  input_list_t *input_list;

  FILE *file = fopen("data/SNP.csv", "r");

  if (file == NULL) {
    perror("Error opening file");
    return 1;
  }
  input_list = read_input(file);
  fclose(file);

  if (input_list == NULL) {
    perror("Error reading input");
    exit(2);
  }

  N = input_list->size;

  // Allocate arrays for X and Y on the CPU. This memory is only usable on the
  // CPU
  bs_inputs_t *CPU_blackScholes_inputs = input_list->list;
  option_price_t CPU_prices[N];

  // GPU
  bs_inputs_t *GPU_blackScholes_inputs;
  option_price_t *GPU_prices;

  if (hipMalloc(&GPU_blackScholes_inputs, sizeof(bs_inputs_t) * N) !=
      hipSuccess) {
    fprintf(stderr, "Failed to allocate X array on GPU\n");
    exit(2);
  }

  // Allocate space for the x array on the GPU
  if (hipMalloc(&GPU_prices, sizeof(option_price_t) * N) != hipSuccess) {
    fprintf(stderr, "Failed to allocate X array on GPU\n");
    exit(2);
  }

  // Copy the cpu's x array to the gpu with hipMemcpy
  if (hipMemcpy(GPU_blackScholes_inputs, CPU_blackScholes_inputs,
                 sizeof(bs_inputs_t) * N,
                 hipMemcpyHostToDevice) != hipSuccess) {
    fprintf(stderr, "Failed to copy X to the GPU\n");
  }

  // Calculate the number of blocks to run, rounding up to include all threads
  size_t blocks = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
  // Start timing the performance
  size_t start_time = time_micros();
  // Run the saxpy kernel
  pricer<<<blocks, THREADS_PER_BLOCK>>>(GPU_blackScholes_inputs, GPU_prices);

  // Wait for the kernel to finish
  if (hipDeviceSynchronize() != hipSuccess) {
    fprintf(stderr, "CUDA Error: %s\n",
            hipGetErrorString(hipPeekAtLastError()));
  }
  // Calculate the elapsed time in miliseconds
  size_t elapsed_time = time_micros() - start_time;
  double seconds = (double)elapsed_time / 1000000;
  double computing_rate = (double)N / seconds;
  printf("Number of options: %d\n", N);
  printf("Total computation time: %lu\u03BCs\n", elapsed_time);
  printf("Computation rate: %.2lf options per second\n", computing_rate);
  // Copy the y array back from the gpu to the cpu
  if (hipMemcpy(CPU_prices, GPU_prices, sizeof(option_price_t) * N,
                 hipMemcpyDeviceToHost) != hipSuccess) {
    fprintf(stderr, "Failed to copy Y from the GPU\n");
  }

  FILE *output_file = fopen("prices_output.csv", "w");

  if (output_file == NULL) {
    perror("Error opening file");
    return 1;
  }
  fprintf(output_file, "call_prices,put_prices");
  // Print the updated y array
  for (int i = 0; i < N; i++) {
    // printf("%d: %f\n",i,CPU_prices[i]);
    fprintf(output_file, "%f,%f\n", CPU_prices[i].call_price,
            CPU_prices[i].put_price);
  }

  fclose(output_file);

  free(input_list);
  hipFree(GPU_prices);
  hipFree(GPU_blackScholes_inputs);
  free(CPU_blackScholes_inputs);
  return 0;
}
